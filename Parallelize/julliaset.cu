#include "thirdparty/book.h"
#include "thirdparty/cpu_bitmap.h"
#define DIM 1000
struct hipComplex
{
    float r;
    float i;
    hipComplex( float a, float b ) : r(a), i(b) {}
    __device__ float magnitude2(void)
    {
        return r*r + i*i;
    }
    __device__ hipComplex operator*(const hipComplex&a){
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    __device__ hipComplex operator+(const hipComplex&a){
        return hipComplex(r+a.r,i+a.i);
    }
    __device__ int julia(int x, int y){
        const float scale = 1.5;
        float jx = scale * float(DIM/2-x)/(DIM/2);
        float jy = scale * (float)(DIM/2 - y)/(DIM/2);
        hipComplex c(-0.8, 0.156);
        hipComplex a(jx, jy);
        
    }
};




int main(){
    return 0;
}