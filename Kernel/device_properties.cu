#include"./thirdparty/book.h"
#include <iostream>

int main(void){
    hipDeviceProp_t prop;
    int dev;
    hipGetDevice( &dev ) ;
    printf( "ID of current CUDA device: %d\n", dev );
    memset(&prop, 0, sizeof(hipDeviceProp_t))  
}